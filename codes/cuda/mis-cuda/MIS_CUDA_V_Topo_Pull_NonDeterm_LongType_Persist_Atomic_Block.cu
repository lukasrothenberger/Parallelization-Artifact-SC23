#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef int flag_t;
typedef unsigned long long data_type;
static const int ThreadsPerBlock = 512;

#include "indigo_mis_vertex_cuda.h"

static __global__ void init(data_type* const priority, flag_t* const status, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size)
  {
    priority[v] = ((unsigned long)hash(v + 712313887)) | ((unsigned long)hash(v + 683067839) << (sizeof (unsigned int) * 8));
    status[v] = undecided;
  }
}

static __global__ void mis(const ECLgraph g, const data_type* const priority, flag_t* const status, flag_t* const goagain)
{
  // go over all the nodes
  int tid = blockIdx.x;
  for (int v = tid; v < g.nodes; v += gridDim.x) {

    if (__syncthreads_or((threadIdx.x == 0) && (atomicRead(&status[v]) == undecided))) {
      int i = g.nindex[v];
      // try to find a non-excluded neighbor whose priority is higher
      if (threadIdx.x == 0) {
        while ((i < g.nindex[v + 1]) && ((atomicRead(&status[g.nlist[i]]) == excluded) || (priority[v] > priority[g.nlist[i]]) || ((priority[v] == priority[g.nlist[i]]) && (v > g.nlist[i])))) {
          i++;
        }
      }
      if (__syncthreads_or((threadIdx.x == 0) && (i < g.nindex[v + 1]))) {
        // found such a neighbor -> check if neighbor is included
        if (__syncthreads_or((threadIdx.x == 0) && (atomicRead(&status[g.nlist[i]]) == included))) {
          // found included neighbor -> exclude self
          if (threadIdx.x == 0) {
            atomicWrite(&status[v], excluded);
          }
        } else { // v still undecided, go again
        if (threadIdx.x == 0) {
          atomicWrite(goagain, 1);
        }
      }
    } else {
      // no such neighbor -> self is "included"
      if (threadIdx.x == 0) {
        atomicWrite(&status[v], included);
      }
    }
  }
}
}

static double GPUmis_vertex(const ECLgraph& g, data_type* const priority, int* const status)
{
flag_t* d_goagain;
if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
data_type* d_priority;
if (hipSuccess != hipMalloc((void **)&d_priority, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_priority\n");
flag_t* d_status;
if (hipSuccess != hipMalloc((void **)&d_status, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status\n");

const int ThreadsBound = GPUinfo(0, false);
const int blocks = ThreadsBound / ThreadsPerBlock;

init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_priority, d_status, g.nodes);

timeval beg, end;
gettimeofday(&beg, NULL);

flag_t goagain;
int iter = 0;
do {
  iter++;
  hipMemset(d_goagain, 0, sizeof(flag_t));

  mis<<<blocks, ThreadsPerBlock>>>(g, d_priority, d_status, d_goagain);

  if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of goagain from device failed\n");
} while (goagain);

hipDeviceSynchronize();
gettimeofday(&end, NULL);
const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;

CheckCuda();
if (hipSuccess != hipMemcpy(status, d_status, g.nodes * sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of status from device failed\n");

// determine and print set size
int cnt = 0;
for (int v = 0; v < g.nodes; v++) {
  if (status[v] == included) cnt++;
}
printf("iterations: %d,  elements in set: %d (%.1f%%)\n", iter, cnt, 100.0 * cnt / g.nodes);

hipFree(d_goagain);
hipFree(d_status);
hipFree(d_priority);
return runtime;
}
