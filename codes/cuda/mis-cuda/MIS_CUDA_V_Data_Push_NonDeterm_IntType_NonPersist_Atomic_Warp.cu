#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef int flag_t;
typedef int data_type;
static const int ThreadsPerBlock = 512;
static const int WarpSize = 32;

#include "mis_vertex_cuda.h"

static __global__ void init(data_type* const priority, flag_t* const status, const int size, int* const wl1, int* const wlsize)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size)
  {
    priority[v] = hash(v + 712313887);
    status[v] = undecided;

    // initialize worklist
    wl1[v] = v;
  }
  if (v == 0) {
    *wlsize = size;
  }
}

static __global__ void mis(const ECLgraph g, const data_type* const priority, flag_t* const status, const int* const wl1, const int wl1size, int* const wl2, int* const wl2size)
{
  const int lane = threadIdx.x % WarpSize;
  // go over all nodes in worklist
  int w = (threadIdx.x + blockIdx.x * ThreadsPerBlock) / WarpSize;
  if (w < wl1size) {

    int v = wl1[w];
    if (__any_sync(~0, (lane == 0) && (atomicRead(&status[v]) == undecided))) {
      int i = g.nindex[v];
      // try to find a non-excluded neighbor whose priority is higher
      if (lane == 0) {
        while ((i < g.nindex[v + 1]) && ((atomicRead(&status[g.nlist[i]]) == excluded) || (priority[v] > priority[g.nlist[i]]) || ((priority[v] == priority[g.nlist[i]]) && (v > g.nlist[i])))) {
          i++;
        }
      }
      if (__any_sync(~0, (lane == 0) && (i < g.nindex[v + 1]))) {
        // found such a neighbor -> status still unknown
        if (lane == 0) {
          wl2[atomicAdd(wl2size, 1)] = v;
        }
      } else {
        // no such neighbor -> all neighbors are "excluded" and v is "included"
        if (lane == 0) {
          atomicWrite(&status[v], included);
        }
        for (int j = g.nindex[v] + lane; j < g.nindex[v + 1]; j += WarpSize) {
          atomicWrite(&status[g.nlist[j]], excluded);
        }
      }
    }
  }
}

static double GPUmis_vertex(const ECLgraph& g, data_type* const priority, int* const status)
{
  data_type* d_priority;
  if (hipSuccess != hipMalloc((void **)&d_priority, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_priority\n");
  flag_t* d_status;
  if (hipSuccess != hipMalloc((void **)&d_status, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status\n");

  int* d_wl1;
  if (hipSuccess != hipMalloc((void **)&d_wl1, g.nodes * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1\n");
  int* d_wl1size;
  if (hipSuccess != hipMalloc((void **)&d_wl1size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1size\n");
  int* d_wl2;
  if (hipSuccess != hipMalloc((void **)&d_wl2, g.nodes * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2\n");
  int* d_wl2size;
  if (hipSuccess != hipMalloc((void **)&d_wl2size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2size\n");
  int wlsize;


  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_priority, d_status, g.nodes, d_wl1, d_wl1size);

  if (hipSuccess != hipMemcpy(&wlsize, d_wl1size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device d_wl1size failed\n");

  timeval beg, end;
  gettimeofday(&beg, NULL);

  int iter = 0;
  do {
    iter++;
    hipMemset(d_wl2size, 0, sizeof(int));
    const int blocks = ((long)wlsize * WarpSize + ThreadsPerBlock - 1) / ThreadsPerBlock;

    mis<<<blocks, ThreadsPerBlock>>>(g, d_priority, d_status, d_wl1, wlsize, d_wl2, d_wl2size);

    if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) { fprintf(stderr, "ERROR: copying of wlsize from device failed\n"); break; }
    std::swap(d_wl1, d_wl2);
    std::swap(d_wl1size, d_wl2size);
  } while (wlsize > 0);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;

  CheckCuda();
  if (hipSuccess != hipMemcpy(status, d_status, g.nodes * sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of status from device failed\n");

  // determine and print set size
  int cnt = 0;
  for (int v = 0; v < g.nodes; v++) {
    if (status[v] == included) cnt++;
  }
  printf("iterations: %d,  elements in set: %d (%.1f%%)\n", iter, cnt, 100.0 * cnt / g.nodes);

  hipFree(d_status);
  hipFree(d_priority);
  hipFree(d_wl1);
  hipFree(d_wl1size);
  hipFree(d_wl2);
  hipFree(d_wl2size);
  return runtime;
}
