#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef cuda::atomic<int> flag_t;
typedef int data_type;
static const int ThreadsPerBlock = 512;

#include "mis_edge_cuda.h"

static __global__ void init(data_type* const priority, flag_t* const status, flag_t* const status_n, flag_t* const lost, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size)
  {
    priority[v] = hash(v + 712313887);
    status[v] = undecided;
    status_n[v] = undecided;
    lost[v] = 0;
  }
}

static __global__ void mis(const ECLgraph g, const int* const sp, const data_type* const priority, flag_t* const status, flag_t* const status_n, flag_t* const lost)
{
  // go over all edges
  int tid = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  for (int e = tid; e < g.edges; e += gridDim.x * ThreadsPerBlock) {

    const int src = sp[e];
    const int dst = g.nlist[e];
    const int srcStatus = status[src].load();
    const int dstStatus = status[dst].load();

    // if one is included, exclude the other
    if (srcStatus == included) {
      status_n[dst].store(excluded);
    }
    else if (dstStatus == included) {
      status_n[src].store(excluded);
    } else if (srcStatus == undecided && dstStatus == undecided) {
      // if both undecided -> mark lower as lost
      if (priority[src] < priority[dst]) {
        lost[src].store(1);
      } else {
        lost[dst].store(1);
      }
    }
  }
}

static __global__ void mis_vertex_pass(const ECLgraph g, const int* const sp, flag_t* const status, flag_t* const status_n, flag_t* const lost, flag_t* const goagain)
{
  // go over all edges
  int tid = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  for (int e = tid; e < g.edges; e += gridDim.x * ThreadsPerBlock) {

    const int src = sp[e];
    const int dst = g.nlist[e];
    const int srcStatus = status[src].load();
    const int dstStatus = status[dst].load();

    // if v didn't lose
    // if src won
    if (lost[src] == 0) {
      if (srcStatus == undecided) {
        // and is undecided -> include
        status_n[src].store(included);
      }
    }
    // if dst won
    if (lost[dst] == 0) {
      if (dstStatus == undecided) {
        // and is undecided -> include
        status_n[dst].store(included);
      }
    }
    // if either is still undecided, goagain
    if (srcStatus == undecided || dstStatus == undecided) {
      *goagain = 1;
    }
  }
}

static __global__ void mis_last_pass(flag_t* const status, const int size)
{
  int tid = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  for (int w = tid; w < size; w += gridDim.x * ThreadsPerBlock) {
    if (status[w] == undecided)
    {
      status[w] = included;
    }
  }
}

static double GPUmis_edge(const ECLgraph& g, const int* const sp, data_type* const priority, int* const status)
{
  flag_t* d_goagain;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  data_type* d_priority;
  if (hipSuccess != hipMalloc((void **)&d_priority, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_priority\n");
  flag_t* d_status;
  if (hipSuccess != hipMalloc((void **)&d_status, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status\n");
  flag_t* d_lost;
  if (hipSuccess != hipMalloc((void **)&d_lost, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_lost\n");
  flag_t* d_status_new;
  if (hipSuccess != hipMalloc((void **)&d_status_new, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status_new\n");

  const int ThreadsBound = GPUinfo(0, false);
  const int blocks = ThreadsBound / ThreadsPerBlock;

  init<<<(g.edges + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_priority, d_status, d_status_new, d_lost, g.nodes);

  timeval beg, end;
  gettimeofday(&beg, NULL);

  flag_t goagain;
  int iter = 0;
  do {
    iter++;
    hipMemset(d_goagain, 0, sizeof(flag_t));

    // edge pass
    mis<<<blocks, ThreadsPerBlock>>>(g, sp, d_priority, d_status, d_status_new, d_lost);

    if (hipSuccess != hipMemcpy(d_status, d_status_new, g.nodes * sizeof(flag_t), hipMemcpyDeviceToDevice)) fprintf(stderr, "ERROR: copying of d_status_new to d_status on device failed\n");
    // vertex pass
    mis_vertex_pass<<<blocks, ThreadsPerBlock>>>(g, sp, d_status, d_status_new, d_lost, d_goagain);

    hipMemset(d_lost, 0, g.nodes * sizeof(flag_t));
    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
    if (hipSuccess != hipMemcpy(d_status, d_status_new, g.nodes * sizeof(flag_t), hipMemcpyDeviceToDevice)) fprintf(stderr, "ERROR: copying of d_status_new to d_status on device failed\n");
  } while (goagain);

  // include all remaining nodes that have no edges
  mis_last_pass<<<blocks, ThreadsPerBlock>>>(d_status, g.nodes);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;

  CheckCuda();
  if (hipSuccess != hipMemcpy(status, d_status, g.nodes * sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of status from device failed\n");

  // determine and print set size
  int cnt = 0;
  for (int v = 0; v < g.nodes; v++) {
    if (status[v] == included) cnt++;
  }
  printf("iterations: %d,  elements in set: %d (%.1f%%)\n", iter, cnt, 100.0 * cnt / g.nodes);

  hipFree(d_status_new);
  hipFree(d_status);
  hipFree(d_priority);
  hipFree(d_lost);
  hipFree(d_goagain);
  return runtime;
}
