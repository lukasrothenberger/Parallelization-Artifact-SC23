#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef cuda::atomic<int> flag_t;
typedef cuda::atomic<unsigned long long> data_type;
typedef unsigned long long basic_t;
static const int ThreadsPerBlock = 512;

#include "indigo_bfs_edge_cuda.h"

static __global__ void init(const int src, data_type* const dist, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    const data_type temp = (v == src) ? 0 : maxval;
    dist[v].store(temp);
  }
}

static __global__ void bfs(const ECLgraph g, const int* const sp, data_type* const dist, flag_t* const goagain)
{
  int e = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (e < g.edges) {

    const int src = sp[e];
    const int dst = g.nlist[e];
    const data_type s = dist[src].load();

    if (s != maxval) {
      const data_type new_dist = s + 1;
      if (dist[dst].fetch_min(new_dist) > new_dist) {
        *goagain = 1;
      }
    }
  }
}

static double GPUbfs_edge(const int src, const ECLgraph& g, basic_t* const dist, const int* const sp)
{
  flag_t* d_goagain;
  data_type* d_dist;
  int* d_sp;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  if (hipSuccess != hipMalloc((void **)&d_dist, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_dist\n");
  if (hipSuccess != hipMalloc((void **)&d_sp, sizeof(int) * g.edges)) {fprintf(stderr, "ERROR: could not allocate d_sp\n"); exit(-1);}
  hipMemcpy(d_sp, sp, sizeof(int) * g.edges, hipMemcpyHostToDevice);

  const int blocks = (g.edges + ThreadsPerBlock - 1) / ThreadsPerBlock;

  timeval start, end;
  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(src, d_dist, g.nodes);

  // iterate until no more changes
  int goagain;
  int iter = 0;
  gettimeofday(&start, NULL);

  do {
    iter++;
    goagain = 0;
    if (hipSuccess != hipMemcpy(d_goagain, &goagain, sizeof(int), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");

    bfs<<<blocks, ThreadsPerBlock>>>(g, d_sp, d_dist, d_goagain);

    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  } while (goagain);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("iterations: %d\n", iter);

  CheckCuda();
  if (hipSuccess != hipMemcpy(dist, d_dist, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of dist from device failed\n");

  hipFree(d_goagain);
  hipFree(d_dist);
  return runtime;
}
