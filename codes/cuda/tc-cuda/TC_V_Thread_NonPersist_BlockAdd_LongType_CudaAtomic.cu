#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef cuda::atomic<unsigned long long> data_type;
typedef unsigned long long basic_t;
static const int ThreadsPerBlock = 512;
#include "indigo_tc_vertex_cuda.h"
static __global__ void d_triCounting(data_type* g_count, const int nodes, const int* const __restrict__ nindex, const int* const __restrict__ nlist)
{
  __shared__ int count;
  if (threadIdx.x == 0) count = 0;
  __syncthreads();
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < nodes) {
    const int beg1 = nindex[v];
    const int end1 = nindex[v + 1];
    int start1 = end1;
    while ((beg1 < start1) && (v < nlist[start1 - 1])) start1--;
    for (int j = start1; j < end1; j++) {
      const int u = nlist[j];
      const int beg2 = nindex[u];
      const int end2 = nindex[u + 1];
      int start2 = end2;
      while ((beg2 < start2) && (u < nlist[start2 - 1])) start2--;
      atomicAdd_block(&count, (basic_t)d_common(j + 1, end1, start2, end2, nlist));
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) (*g_count) += count;
}
static double GPUtc_vertex(basic_t &count, const int nodes, const int* const nindex, const int* const nlist)
{
  data_type* d_count;
  if (hipSuccess != hipMalloc((void **)&d_count, sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  timeval start, end;
  const int blocks = (nodes + ThreadsPerBlock - 1) / ThreadsPerBlock;
  count = 0;
  gettimeofday(&start, NULL);
  if (hipSuccess != hipMemcpy(d_count, &count, sizeof(data_type), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");
  d_triCounting<<<blocks, ThreadsPerBlock>>>(d_count, nodes, nindex, nlist);
  if (hipSuccess != hipMemcpy(&count, d_count, sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  hipFree(d_count);
  return (end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0);
}
