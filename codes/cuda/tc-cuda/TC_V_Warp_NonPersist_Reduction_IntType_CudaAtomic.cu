#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef cuda::atomic<int> data_type;
typedef int basic_t;
static const int WS = 32;
static const int ThreadsPerBlock = 512;
#include "tc_vertex_cuda.h"
static __global__ void d_triCounting(data_type* g_count, const int nodes, const int* const __restrict__ nindex, const int* const __restrict__ nlist)
{
  __shared__ int s_buffer[WS];
  const int lane = threadIdx.x % WS;
  const int warp = threadIdx.x / WS;
  basic_t count = 0;
  const int v = (threadIdx.x + blockIdx.x * ThreadsPerBlock) / WS;
  if (v < nodes) {
    const int beg1 = nindex[v];
    const int end1 = nindex[v + 1];
    int start1 = end1;
    while ((beg1 < start1) && (v < nlist[start1 - 1])) start1--;
    for (int j = start1 + threadIdx.x % WS; j < end1; j += WS){
      const int u = nlist[j];
      const int beg2 = nindex[u];
      const int end2 = nindex[u + 1];
      int start2 = end2;
      while ((beg2 < start2) && (u < nlist[start2 - 1])) start2--;
      count += (basic_t)d_common(j + 1, end1, start2, end2, nlist);
    }
  }
  // warp reduction
  count += __shfl_down_sync(~0, count, 16);
  count += __shfl_down_sync(~0, count, 8);
  count += __shfl_down_sync(~0, count, 4);
  count += __shfl_down_sync(~0, count, 2);
  count += __shfl_down_sync(~0, count, 1);
  if (lane == 0) s_buffer[warp] = count;
  __syncthreads();
  // block reduction
  if (warp == 0) {
    int val = s_buffer[lane];
    val += __shfl_down_sync(~0, val, 16);
    val += __shfl_down_sync(~0, val, 8);
    val += __shfl_down_sync(~0, val, 4);
    val += __shfl_down_sync(~0, val, 2);
    val += __shfl_down_sync(~0, val, 1);
    if (lane == 0) (*g_count) += val;
  }
}
static double GPUtc_vertex(basic_t &count, const int nodes, const int* const nindex, const int* const nlist)
{
  data_type* d_count;
  if (hipSuccess != hipMalloc((void **)&d_count, sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  timeval start, end;
  const int blocks = ((long)nodes * WS + ThreadsPerBlock - 1) / ThreadsPerBlock;
  count = 0;
  gettimeofday(&start, NULL);
  if (hipSuccess != hipMemcpy(d_count, &count, sizeof(data_type), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");
  d_triCounting<<<blocks, ThreadsPerBlock>>>(d_count, nodes, nindex, nlist);
  if (hipSuccess != hipMemcpy(&count, d_count, sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  hipFree(d_count);
  return (end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0);
}
