#include "hip/hip_runtime.h"
#include <cuda/atomic>

typedef cuda::atomic<int> flag_t;
typedef cuda::atomic<int> data_type;
typedef int basic_t;
#include "cc_vertex_cuda.h"

static const int ThreadsPerBlock = 512;
static const int WarpSize = 32;

static __global__ void init(data_type* const label, data_type* const label_n, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    label_n[v].store(v);
    label[v].store(v);
  }
}

static __global__ void cc(const ECLgraph g, data_type* const label, data_type* const label_n, flag_t* const goagain)
{
  int v = (threadIdx.x + blockIdx.x * ThreadsPerBlock) / WarpSize;
  if (v < g.nodes) {

    const int beg = g.nindex[v];
    const int end = g.nindex[v + 1];
    const data_type new_label = label[v].load();

    bool updated = false;
    for (int i = beg + threadIdx.x % WarpSize; i < end; i += WarpSize) {
      const int dst = g.nlist[i];
      if (label_n[dst].fetch_min(new_label) > new_label) {
        updated = true;
      }
    }
    if (updated) {
      *goagain = 1;
    }
  }
}

static double GPUcc_vertex(const ECLgraph& g, basic_t* const label)
{
  flag_t* d_goagain;
  data_type* d_label;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  if (hipSuccess != hipMalloc((void **)&d_label, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_label\n");
  data_type* d_label_new;
  if (hipSuccess != hipMalloc((void **)&d_label_new, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_label_new\n");

  const int blocks = ((long)g.nodes * WarpSize + ThreadsPerBlock - 1) / ThreadsPerBlock;

  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_label, d_label_new, g.nodes);

  // iterate until no more changes
  int goagain;
  int iter = 0;
  timeval start, end;
  gettimeofday(&start, NULL);

  do {
    iter++;
    goagain = 0;
    if (hipSuccess != hipMemcpy(d_goagain, &goagain, sizeof(flag_t), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");

    cc<<<blocks, ThreadsPerBlock>>>(g, d_label, d_label_new, d_goagain);

    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
    std::swap(d_label, d_label_new);
  } while (goagain);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("iterations: %d\n", iter);

  CheckCuda();
  if (hipSuccess != hipMemcpy(label, d_label_new, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of label from device failed\n");

  hipFree(d_goagain);
  hipFree(d_label);
  return runtime;
}
