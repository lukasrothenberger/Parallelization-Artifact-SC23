#include "hip/hip_runtime.h"
#include <cuda/atomic>

typedef int flag_t;
typedef int data_type;
typedef int basic_t;

#include "indigo_cc_edge_cuda.h"

static const int ThreadsPerBlock = 512;

static __global__ void init(data_type* const label, const ECLgraph g, int* const wl1, int* const wlsize)
{
  // initialize label array
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < g.nodes) {
    label[v] = v;
    for (int i = g.nindex[v]; i < g.nindex[v + 1]; i++) {
      wl1[i] = i;
    }
  }
  if (v == 0) {
    *wlsize = g.edges;
  }
}

static __global__ void cc_edge_data(const ECLgraph g, const int* const sp, data_type* const label, const int* const wl1, const int wl1size, int* const wl2, int* const wl2size, const int iter, int* const time)
{
  for (int idx = threadIdx.x + blockIdx.x * ThreadsPerBlock; idx < wl1size; idx += gridDim.x * ThreadsPerBlock) {
    const int e = wl1[idx];
    const int src = sp[e];
    const int dst = g.nlist[e];
    const data_type new_label = atomicRead(&label[src]);

    if (atomicMin(&label[dst], new_label) > new_label) {
      for (int j = g.nindex[dst]; j < g.nindex[dst + 1]; j++) {
        if (atomicMax(&time[j], iter) != iter) {
          wl2[atomicAdd(wl2size, 1)] = j;
        }
      }
    }
  }
}

static double GPUcc_edge(const ECLgraph& g, basic_t* const label, const int* const sp)
{
  data_type* d_label;
  if (hipSuccess != hipMalloc((void **)&d_label, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_label\n");
  int* d_wl1;
  if (hipSuccess != hipMalloc((void **)&d_wl1, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1\n");
  int* d_wl1size;
  if (hipSuccess != hipMalloc((void **)&d_wl1size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1size\n");

  int* d_wl2;
  if (hipSuccess != hipMalloc((void **)&d_wl2, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2\n");
  int* d_wl2size;
  if (hipSuccess != hipMalloc((void **)&d_wl2size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2size\n");

  int* d_time;
  if (hipSuccess != hipMalloc((void **)&d_time, sizeof(int) * g.edges)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  hipMemset(d_time, 0, sizeof(int) * g.edges);

  int* d_sp;
  if (hipSuccess != hipMalloc((void **)&d_sp, sizeof(int) * g.edges)) {fprintf(stderr, "ERROR: could not allocate d_sp\n"); exit(-1);}
  hipMemcpy(d_sp, sp, sizeof(int) * g.edges, hipMemcpyHostToDevice);

  int wlsize;
  const int ThreadsBound = GPUinfo(0);
  const int blocks = ThreadsBound / ThreadsPerBlock;

  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_label, g, d_wl1, d_wl2size);

  if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device failed\n");
  if (hipSuccess != hipMemcpy(d_wl1size, &wlsize, sizeof(int), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of wl1size to device failed\n");
  // iterate until no more changes
  int iter = 0;

  timeval start, end;
  gettimeofday(&start, NULL);

  do {
    iter++;
    hipMemset(d_wl2size, 0, sizeof(int));

    cc_edge_data<<<blocks, ThreadsPerBlock>>>(g, d_sp, d_label, d_wl1, wlsize, d_wl2, d_wl2size, iter, d_time);

    if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device failed\n");
    std::swap(d_wl1, d_wl2);
    std::swap(d_wl1size, d_wl2size);
  } while (wlsize > 0);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  CheckCuda();
  printf("iterations: %d\n", iter);

  if (hipSuccess != hipMemcpy(label, d_label, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of label from device failed\n");

  hipFree(d_label);
  hipFree(d_wl1);
  hipFree(d_wl1size);
  hipFree(d_wl2);
  hipFree(d_wl2size);
  return runtime;
}
