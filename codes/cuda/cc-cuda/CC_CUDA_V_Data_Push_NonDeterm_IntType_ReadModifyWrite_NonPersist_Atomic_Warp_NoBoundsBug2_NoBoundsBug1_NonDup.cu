#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef int flag_t;
typedef int data_type;
typedef int basic_t;
static const int ThreadsPerBlock = 512;
static const int WarpSize = 32;

#include "cc_vertex_cuda.h"

static __global__ void init(data_type* const label, const int size, const ECLgraph g, int* const wl1, int* const wlsize)
{
  // initialize label array
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    label[v] = v;
    wl1[v] = v;
  }
  // initialize worklist
  if (v == 0) {
    // wl1[0] = 0;
    *wlsize = size;
  }
}

static __global__ void cc_vertex_data(const ECLgraph g, data_type* const label, const int* const wl1, const int wl1size, int* const wl2, int* const wl2size, const int iter, int* const time)
{
  int idx = (threadIdx.x + blockIdx.x * ThreadsPerBlock) / WarpSize;
  if (idx < wl1size) {
    const int src = wl1[idx];
    const data_type new_label = atomicRead(&label[src]);
    const int beg = g.nindex[src];
    const int end = g.nindex[src + 1];

    for (int i = beg + threadIdx.x % WarpSize; i < end; i += WarpSize) {
      const int dst = g.nlist[i];

      if (atomicMin(&label[dst], new_label) > new_label) {
        if (atomicMax(&time[dst], iter) != iter) {
          wl2[atomicAdd(wl2size, 1)] = dst;
        }
      }
    }
    __syncwarp();
  }
}
static double GPUcc_vertex(const ECLgraph& g, basic_t* const label)
{
  data_type* d_label;
  if (hipSuccess != hipMalloc((void **)&d_label, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_label\n");
  int* d_wl1;
  if (hipSuccess != hipMalloc((void **)&d_wl1, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1\n");
  int* d_wl1size;
  if (hipSuccess != hipMalloc((void **)&d_wl1size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1size\n");
  int* d_wl2;
  if (hipSuccess != hipMalloc((void **)&d_wl2, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2\n");
  int* d_wl2size;
  if (hipSuccess != hipMalloc((void **)&d_wl2size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2size\n");
  int* d_time;
  if (hipSuccess != hipMalloc((void **)&d_time, sizeof(int) * g.nodes)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  hipMemset(d_time, 0, sizeof(int) * g.nodes);
  int wlsize;

  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_label, g.nodes, g, d_wl1, d_wl2size);

  if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device failed\n");
  if (hipSuccess != hipMemcpy(d_wl1size, &wlsize, sizeof(int), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of wl1size to device failed\n");
  // iterate until no more changes
  int iter = 0;

  timeval start, end;
  gettimeofday(&start, NULL);

  do {
    iter++;
    hipMemset(d_wl2size, 0, sizeof(int));
    const int blocks = ((long)wlsize * WarpSize + ThreadsPerBlock - 1) / ThreadsPerBlock;

    cc_vertex_data<<<blocks, ThreadsPerBlock>>>(g, d_label, d_wl1, wlsize, d_wl2, d_wl2size, iter, d_time);

    if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device failed\n");
    std::swap(d_wl1, d_wl2);
    std::swap(d_wl1size, d_wl2size);
  } while (wlsize > 0);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);

  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  CheckCuda();
  printf("iterations: %d\n", iter);

  if (hipSuccess != hipMemcpy(label, d_label, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of label from device failed\n");

  hipFree(d_label);
  hipFree(d_wl1);
  hipFree(d_wl1size);
  hipFree(d_wl2);
  hipFree(d_wl2size);
  hipFree(d_time);
  return runtime;
}
