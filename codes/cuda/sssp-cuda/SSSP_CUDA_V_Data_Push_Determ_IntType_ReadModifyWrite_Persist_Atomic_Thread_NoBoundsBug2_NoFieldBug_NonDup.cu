#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef int flag_t;
typedef int data_type;
typedef int basic_t;
static const int ThreadsPerBlock = 512;

#include "sssp_vertex_cuda.h"

static __global__ void init(const int src, data_type* const dist, data_type* const dist_n, const int size, const ECLgraph g, int* const wl1, int* const wlsize)
{
  // initialize dist array
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    const data_type temp = (v == src) ? 0 : maxval;
    dist_n[v] = temp;
    dist[v] = temp;
  }
  // initialize worklist
  if (v == 0) {
    wl1[0] = src;
    *wlsize = 1;
  }
}

static __global__ void sssp_vertex_data(const ECLgraph g, data_type* const dist, data_type* const dist_n, const int* const wl1, const int wl1size, int* const wl2, int* const wl2size, const int iter, int* const time)
{
  int tid = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  for (int idx = tid; idx < wl1size; idx += gridDim.x * ThreadsPerBlock) {
    const int src = wl1[idx];
    const data_type s = dist[src];
    const int beg = g.nindex[src];
    const int end = g.nindex[src + 1];

    if (s != maxval) {
      for (int i = beg; i < end; i++) {
        const int dst = g.nlist[i];
        const data_type new_dist = s + g.eweight[i];

        if (atomicMin(&dist_n[dst], new_dist) > new_dist) {
          if (atomicMax(&time[dst], iter) < iter) {
            wl2[atomicAdd(wl2size, 1)] = dst;
          }
        }
      }
      atomicMin(&dist_n[src], s);
    }
  }
}
static double GPUsssp_vertex(const int src, const ECLgraph& g, basic_t* const dist)
{
  data_type* d_dist;
  if (hipSuccess != hipMalloc((void **)&d_dist, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_dist\n");
  data_type* d_dist_new;
  if (hipSuccess != hipMalloc((void **)&d_dist_new, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_dist_new\n");
  int* d_wl1;
  if (hipSuccess != hipMalloc((void **)&d_wl1, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1\n");
  int* d_wl1size;
  if (hipSuccess != hipMalloc((void **)&d_wl1size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1size\n");
  int* d_wl2;
  if (hipSuccess != hipMalloc((void **)&d_wl2, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2\n");
  int* d_wl2size;
  if (hipSuccess != hipMalloc((void **)&d_wl2size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2size\n");
  int* d_time;
  if (hipSuccess != hipMalloc((void **)&d_time, sizeof(int) * g.nodes)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  hipMemset(d_time, 0, sizeof(int) * g.nodes);
  int wlsize;
  const int ThreadsBound = GPUinfo(0);
  const int blocks = ThreadsBound / ThreadsPerBlock;

  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(src, d_dist, d_dist_new, g.nodes, g, d_wl1, d_wl2size);

  if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device failed\n");
  if (hipSuccess != hipMemcpy(d_wl1size, &wlsize, sizeof(int), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of wl1size to device failed\n");
  // iterate until no more changes
  int iter = 0;

  timeval start, end;
  gettimeofday(&start, NULL);

  do {
    iter++;
    hipMemset(d_wl2size, 0, sizeof(int));

    sssp_vertex_data<<<blocks, ThreadsPerBlock>>>(g, d_dist, d_dist_new, d_wl1, wlsize, d_wl2, d_wl2size, iter, d_time);

    if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device failed\n");
    std::swap(d_wl1, d_wl2);
    std::swap(d_wl1size, d_wl2size);
    std::swap(d_dist, d_dist_new);
  } while (wlsize > 0);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);

  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  CheckCuda();
  printf("iterations: %d\n", iter);

  if (hipSuccess != hipMemcpy(dist, d_dist_new, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of dist from device failed\n");

  hipFree(d_dist);
  hipFree(d_wl1);
  hipFree(d_wl1size);
  hipFree(d_wl2);
  hipFree(d_wl2size);
  hipFree(d_time);
  return runtime;
}
