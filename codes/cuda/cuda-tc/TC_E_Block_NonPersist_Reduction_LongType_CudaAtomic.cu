#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef cuda::atomic<unsigned long long> data_type;
typedef unsigned long long basic_t;
static const int WS = 32;
static const int ThreadsPerBlock = 512;
#include "indigo_tc_edge_cuda.h"
static __global__ void d_triCounting(data_type* g_count, const int edges, const int* const __restrict__ nindex, const int* const __restrict__ nlist, const int* const sp)
{
  __shared__ int s_buffer[WS];
  const int lane = threadIdx.x % WS;
  const int warp = threadIdx.x / WS;
  basic_t count = 0;
  const int e = blockIdx.x;
  if (e < edges) {
    const int src = sp[e];
    const int dst = nlist[e];
    if (src > dst) {
      const int beg1 = nindex[dst];
      const int end1 = nindex[dst + 1];
      for (int i = beg1 + threadIdx.x; i < end1 && nlist[i] < dst; i += ThreadsPerBlock){
        const int u = nlist[i];
        int beg2 = nindex[src];
        int end2 = nindex[src + 1];
        if (d_find(u, beg2, end2, nlist)) count++;
      }
    }
  }
  // warp reduction
  count += __shfl_down_sync(~0, count, 16);
  count += __shfl_down_sync(~0, count, 8);
  count += __shfl_down_sync(~0, count, 4);
  count += __shfl_down_sync(~0, count, 2);
  count += __shfl_down_sync(~0, count, 1);
  if (lane == 0) s_buffer[warp] = count;
  __syncthreads();
  // block reduction
  if (warp == 0) {
    int val = s_buffer[lane];
    val += __shfl_down_sync(~0, val, 16);
    val += __shfl_down_sync(~0, val, 8);
    val += __shfl_down_sync(~0, val, 4);
    val += __shfl_down_sync(~0, val, 2);
    val += __shfl_down_sync(~0, val, 1);
    if (lane == 0) (*g_count) += val;
  }
}
static double GPUtc_edge(basic_t &count, const int edges, const int* const nindex, const int* const nlist, const int* const sp)
{
  data_type* d_count;
  if (hipSuccess != hipMalloc((void **)&d_count, sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  timeval start, end;
  const int blocks = edges;
  count = 0;
  gettimeofday(&start, NULL);
  if (hipSuccess != hipMemcpy(d_count, &count, sizeof(data_type), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");
  d_triCounting<<<blocks, ThreadsPerBlock>>>(d_count, edges, nindex, nlist, sp);
  if (hipSuccess != hipMemcpy(&count, d_count, sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  hipFree(d_count);
  return (end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0);
}
