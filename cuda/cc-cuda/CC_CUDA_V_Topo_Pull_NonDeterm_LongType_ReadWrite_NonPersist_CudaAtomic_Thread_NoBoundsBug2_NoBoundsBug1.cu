#include "hip/hip_runtime.h"
#include <cuda/atomic>

typedef cuda::atomic<int> flag_t;
typedef cuda::atomic<unsigned long long> data_type;
typedef unsigned long long basic_t;

#include "indigo_cc_vertex_cuda.h"

static const int ThreadsPerBlock = 512;

static __global__ void init(data_type* const label, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    label[v].store(v);
  }
}

static __global__ void cc(const ECLgraph g, data_type* const label, flag_t* const goagain)
{
  int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < g.nodes) {

    const int beg = g.nindex[v];
    const int end = g.nindex[v + 1];
    data_type d = label[v].load();

    for (int i = beg; i < end; i++) {
      const int src = g.nlist[i];
      const data_type new_label  = label[src].load();
      if (new_label < d) {
        d = new_label.load();
        *goagain = 1;
      }
    }
    label[v].store(d);
  }
}

static double GPUcc_vertex(const ECLgraph& g, basic_t* const label)
{
  flag_t* d_goagain;
  data_type* d_label;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  if (hipSuccess != hipMalloc((void **)&d_label, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_label\n");

  const int blocks = (g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock;

  init<<<blocks, ThreadsPerBlock>>>(d_label, g.nodes);

  // iterate until no more changes
  int goagain;
  int iter = 0;
  timeval start, end;
  gettimeofday(&start, NULL);

  do {
    iter++;
    goagain = 0;
    if (hipSuccess != hipMemcpy(d_goagain, &goagain, sizeof(flag_t), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");

    cc<<<blocks, ThreadsPerBlock>>>(g, d_label, d_goagain);

    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  } while (goagain);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("iterations: %d\n", iter);

  CheckCuda();
  if (hipSuccess != hipMemcpy(label, d_label, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of label from device failed\n");

  hipFree(d_goagain);
  hipFree(d_label);
  return runtime;
}
