#include "hip/hip_runtime.h"
#include <cuda/atomic>
typedef cuda::atomic<int> flag_t;
typedef cuda::atomic<unsigned long long> data_type;
typedef unsigned long long basic_t;
static const int ThreadsPerBlock = 512;
static const int WarpSize = 32;

#include "indigo_bfs_vertex_cuda.h"

static __global__ void init(const int src, data_type* const dist, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    const data_type temp = (v == src) ? 0 : maxval;
    dist[v].store(temp);
  }
}

static __global__ void bfs(const int iter, const ECLgraph g, data_type* const dist, flag_t* const goagain)
{
  int v = (threadIdx.x + blockIdx.x * ThreadsPerBlock) / WarpSize;
  if (v < g.nodes) {

    const int beg = g.nindex[v];
    const int end = g.nindex[v + 1];
    const data_type s = dist[v].load();

    if (s != maxval) {
      bool updated = false;
      for (int i = beg + threadIdx.x % WarpSize; i < end; i += WarpSize) {
        const int dst = g.nlist[i];
        const data_type new_dist = s + 1;
        if (dist[dst].fetch_min(new_dist) > new_dist) {
          updated = true;
        }
      }
      if (updated) {
        *goagain = 1;
      }
    }
  }
}

static double GPUbfs_vertex(const int src, const ECLgraph& g, basic_t* const dist)
{
  flag_t* d_goagain;
  data_type* d_dist;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  if (hipSuccess != hipMalloc((void **)&d_dist, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_dist\n");

  const int blocks = ((long)g.nodes * WarpSize + ThreadsPerBlock - 1) / ThreadsPerBlock;

  timeval start, end;
  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(src, d_dist, g.nodes);

  // iterate until no more changes
  int goagain;
  int iter = 0;
  gettimeofday(&start, NULL);

  do {
    goagain = 0;
    if (hipSuccess != hipMemcpy(d_goagain, &goagain, sizeof(flag_t), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");

    bfs<<<blocks, ThreadsPerBlock>>>(iter, g, d_dist, d_goagain);
    iter++;

    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  } while (goagain);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("iterations: %d\n", iter);

  CheckCuda();
  if (hipSuccess != hipMemcpy(dist, d_dist, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of dist from device failed\n");

  hipFree(d_goagain);
  hipFree(d_dist);
  return runtime;
}
